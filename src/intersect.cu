#include "hip/hip_runtime.h"
#include "core.h"

__device__ float solve_sphere_line_intersect(float3 p, float3 v, float4 sphere)
{
  float3 cst;
  float delta;
  float2 sol;

  cst.x = SQ(v.x) + SQ(v.y) + SQ(v.z);
  cst.y = 2 * (v.x * (p.x - sphere.x)
	       + v.y * (p.y - sphere.y)
	       + v.z * (p.z - sphere.z));
  cst.z = -2 * (sphere.x * p.x + sphere.y * p.y
    + sphere.z * p.z)
    + SQ(sphere.x) + SQ(sphere.y) + SQ(sphere.z) - SQ(sphere.w)
    + SQ(p.x) + SQ(p.y) + SQ(p.z);
  delta = SQ(cst.y) - (4 * cst.x * cst.z);
  if (delta < 0 || cst.x == 0)
    return (-1);
  sol.x = (-(cst.y) + sqrt(delta)) / (2 * cst.x);
  sol.y = (-(cst.y) - sqrt(delta)) / (2 * cst.x);
  if (sol.x >= 0 && sol.y >= 0)
    return (MIN(sol.x, sol.y));
  return (MAX(sol.x, sol.y));
}

__device__ void intersect(s_var *var, int nb_sphere, const float4 *sphere)
{
  float k;

  var->oc_d = INFINITE;
  for (int i = 0; i < nb_sphere; i++)
  {
    k = solve_sphere_line_intersect(var->c_p, var->c_v, sphere[i]);
    if (k >= 0 && k < var->oc_d)
    {
      var->oc_d = k;
      var->h_i = i;
    }
  }
  k = -var->c_p.z/var->c_v.z;
  if (k >= 0 && k < var->oc_d)
  {
    var->oc_d = k;
    var->h_i = -1;
  }
  if (var->oc_d == INFINITE)
    var->oc_d = -1;
}

__device__ bool is_shadow(s_var *var, int nb_sphere, const float4 *sphere)
{
  float k;
  float k_min = INFINITE;
  int i_min = -1;
  float3 lo_v = vec_inv(var->ol_v);

  for (int i = 0; i < nb_sphere; i++)
  {
    k = solve_sphere_line_intersect(var->l_p, lo_v, sphere[i]);
    if (k >= 0 && k < k_min)
    {
      k_min = k;
      i_min = i;
    }
  }
  if (i_min < 0 && var->h_i >= 0)
    return (false);
  return (i_min ^ var->h_i);
}

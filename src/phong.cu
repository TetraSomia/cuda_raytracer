#include "hip/hip_runtime.h"
#include <math.h>
#include "core.h"

__device__ void phong(s_var *var)
{
  var->oc_v = vec_new_uni(var->h_p, var->c_p);
  var->ol_r = vec_reflec(var->h_n, var->ol_v);
  float coef = vec_dot(var->oc_v, var->ol_r);
  if (coef < 0)
    return;
  if (coef > 1)
    coef = 1;
  coef = powf(coef, PHONG_SIZE);
  //var->coef += coef;
}

#include "hip/hip_runtime.h"
#include <math.h>
#include "core.h"

__device__ float vec_dot(float3 v1, float3 v2)
{
  return (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z);
}

__device__ float3 vec_new(float3 p1, float3 p2)
{
  return ((float3){p2.x - p1.x, p2.y - p1.y, p2.z - p1.z});
}

__device__ float3 vec_new_uni(float3 p1, float3 p2)
{
  float mult;
  float3 v;

  v.x = p2.x - p1.x;
  v.y = p2.y - p1.y;
  v.z = p2.z - p1.z;
  mult = 1.0f / sqrtf(SQ(v.x) + SQ(v.y) + SQ(v.z));
  v.x *= mult;
  v.y *= mult;
  v.z *= mult;
  return (v);
}

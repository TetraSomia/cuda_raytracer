#include "hip/hip_runtime.h"
#include "core.h"

__device__ float solve_sphere_cam_intersect(s_var *var, float4 sphere)
{
  float3 cst;
  float delta;
  float2 sol;

  cst.x = SQ(var->c_v.x) + SQ(var->c_v.y) + SQ(var->c_v.z);
  cst.y = 2 * (var->c_v.x * (var->c_p.x - sphere.x)
	       + var->c_v.y * (var->c_p.y - sphere.y)
	       + var->c_v.z * (var->c_p.z - sphere.z));
  cst.z = -2 * (sphere.x * var->c_p.x + sphere.y * var->c_p.y
    + sphere.z * var->c_p.z)
    + SQ(sphere.x) + SQ(sphere.y) + SQ(sphere.z) - SQ(sphere.w)
    + SQ(var->c_p.x) + SQ(var->c_p.y) + SQ(var->c_p.z);
  delta = SQ(cst.y) - (4 * cst.x * cst.z);
  if (delta < 0 || cst.x == 0)
    return (-1);
  sol.x = (-(cst.y) + sqrt(delta)) / (2 * cst.x);
  sol.y = (-(cst.y) - sqrt(delta)) / (2 * cst.x);
  if (sol.x >= 0 && sol.y >= 0)
    return (MIN(sol.x, sol.y));
  return (MAX(sol.x, sol.y));
}

__device__ void intersect_spheres(s_var *var, uint nb_sphere, const float4 *sphere)
{
  float k;

  var->oc_d = INFINITE;
  for (int i = 0; i < nb_sphere; i++)
  {
    k = solve_sphere_cam_intersect(var, sphere[i]);
    if (k >= 0 && k < var->oc_d)
      var->oc_d = k;
  }
}

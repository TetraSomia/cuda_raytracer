#include "hip/hip_runtime.h"
#include <math.h>
#include "core.h"

__device__ void phong(s_var *var, uint *pix)
{
  var->ol_r = vec_reflec(var->h_n, var->ol_v);
  float coef = vec_dot(var->oc_v, var->ol_r);
  if (coef < 0)
    return;
  coef = powf(coef, PHONG_SIZE);
  uint r,g,b;
  r = ((255 - GET_R(*pix)) / 2) * coef + GET_R(*pix);
  g = ((255 - GET_G(*pix)) / 2) * coef + GET_G(*pix);
  b = ((255 - GET_B(*pix)) / 2) * coef + GET_B(*pix);
  *pix = RGB(r, g, b);
}

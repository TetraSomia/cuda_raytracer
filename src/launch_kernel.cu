#include "core.h"

void launch_kernel(s_data *data)
{
  s_sphere *g_sphere;
  uint *g_pixels;
  s_meta *g_meta;

  hipMalloc(&g_sphere, data->meta.nb_sphere * sizeof(s_sphere));
  hipMalloc(&g_pixels, N * sizeof(uint));
  hipMalloc(&g_meta, sizeof(s_meta));
  hipMemcpy(g_sphere, data->sphere, data->meta.nb_sphere * sizeof(s_sphere), hipMemcpyHostToDevice);
  hipMemcpy(g_pixels, data->pixels, N * sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(g_meta, &data->meta, sizeof(s_meta), hipMemcpyHostToDevice);
  raytrace<<<N/THREADS, THREADS>>>(g_meta, g_sphere, g_pixels);
  hipMemcpy(data->pixels, g_pixels, N * sizeof(uint), hipMemcpyDeviceToHost);
  hipFree(g_sphere);
  hipFree(g_pixels);
  hipFree(g_meta);
}

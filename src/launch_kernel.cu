#include "core.h"

void launch_kernel(s_data *data)
{
  hipMemcpy(data->g_sphere, data->sphere, data->meta.nb_sphere * sizeof(s_sphere), hipMemcpyHostToDevice);
  hipMemcpy(data->g_pixels, data->pixels, N * sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(data->g_meta, &data->meta, sizeof(s_meta), hipMemcpyHostToDevice);
  raytrace<<<N/THREADS, THREADS>>>(data->g_meta, data->g_sphere, data->g_pixels);
  hipMemcpy(data->pixels, data->g_pixels, N * sizeof(uint), hipMemcpyDeviceToHost);
}

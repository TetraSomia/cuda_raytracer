#include "hip/hip_runtime.h"
#include "core.h"

void launch_kernel(s_data *data)
{
  float4 *g_sphere;
  uint *g_pixels;

  hipMalloc(&g_sphere, data->nb_sphere * sizeof(float4));
  hipMalloc(&g_pixels, N * sizeof(uint));
  hipMemcpy(g_sphere, data->sphere, data->nb_sphere * sizeof(float4), hipMemcpyHostToDevice);
  hipMemcpy(g_pixels, data->pixels, N * sizeof(uint), hipMemcpyHostToDevice);
  raytrace<<<N/THREADS, THREADS>>>(data->cam_pos, data->cam_rot, data->nb_sphere, g_sphere, g_pixels);
  hipMemcpy(data->pixels, g_pixels, N * sizeof(uint), hipMemcpyDeviceToHost);
}

#include "hip/hip_runtime.h"
#include "core.h"

void launch_kernel(s_data *data)
{
  float4 *g_sphere;
  uint *g_pixels;
  s_meta *g_meta;

  hipMalloc(&g_sphere, data->meta.nb_sphere * sizeof(float4));
  hipMalloc(&g_pixels, N * sizeof(uint));
  hipMalloc(&g_meta, sizeof(s_meta));
  hipMemcpy(g_sphere, data->sphere, data->meta.nb_sphere * sizeof(float4), hipMemcpyHostToDevice);
  hipMemcpy(g_pixels, data->pixels, N * sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(g_meta, &data->meta, sizeof(s_meta), hipMemcpyHostToDevice);
  raytrace<<<N/THREADS, THREADS>>>(g_meta, g_sphere, g_pixels);
  hipMemcpy(data->pixels, g_pixels, N * sizeof(uint), hipMemcpyDeviceToHost);
  hipFree(g_sphere);
  hipFree(g_pixels);
  hipFree(g_meta);
}

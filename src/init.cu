#include "hip/hip_runtime.h"
#include "core.h"

void init(s_data *data, SDL_Surface *screen)
{
  data->rotation = 0;
  data->pixels = (unsigned int*)screen->pixels;
  data->meta.nb_sphere = 2;
  data->sphere = (float4*)malloc(data->meta.nb_sphere * sizeof(float4));
  data->sphere[0].x = 0;
  data->sphere[0].y = 0;
  data->sphere[0].z = 1;
  data->sphere[0].w = 1;
  data->sphere[1].x = 0;
  data->sphere[1].y = 3;
  data->sphere[1].z = 1;
  data->sphere[1].w = 1;
  data->meta.cam_pos.x = 0;
  data->meta.cam_pos.y = 5;
  data->meta.cam_pos.z = 1;
  data->meta.cam_rot.x = M_PI/2;
  data->meta.cam_rot.y = 0;
  data->meta.cam_rot.z = 0;
  data->meta.light.x = 5;
  data->meta.light.y = 5;
  data->meta.light.z = 5;
}

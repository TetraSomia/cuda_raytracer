#include "core.h"

void init(s_data *data)
{
  data->moved_object = -1;
  data->pixels = (unsigned int*)data->screen->pixels;
  data->meta.nb_sphere = 2;
  data->sphere = (s_sphere*)malloc(data->meta.nb_sphere * sizeof(s_sphere));
  data->sphere[1].pos.x = 0;
  data->sphere[1].pos.y = 0;
  data->sphere[1].pos.z = 1;
  data->sphere[1].r = 1;
  data->sphere[1].color = RGB(255, 0, 0);
  data->sphere[0].pos.x = 0;
  data->sphere[0].pos.y = 3;
  data->sphere[0].pos.z = 1;
  data->sphere[0].r = 1;
  data->sphere[0].color = RGB(0, 255, 0);
  data->meta.cam_pos.x = 0;
  data->meta.cam_pos.y = 5;
  data->meta.cam_pos.z = 1;
  data->meta.cam_rot.x = M_PI/2;
  data->meta.cam_rot.y = 0;
  data->meta.cam_rot.z = 0;
  data->meta.light.x = 5;
  data->meta.light.y = 5;
  data->meta.light.z = 5;
  hipMalloc(&data->g_sphere, data->meta.nb_sphere * sizeof(s_sphere));
  hipMalloc(&data->g_pixels, N * sizeof(uint));
  hipMalloc(&data->g_meta, sizeof(s_meta));
}

#include "hip/hip_runtime.h"
#include "core.h"

void init(s_data *data, SDL_Surface *screen)
{
  data->rotation = 0;
  data->pixels = (unsigned int*)screen->pixels;
  data->meta.nb_sphere = 1;
  data->sphere = (float4*)malloc(data->meta.nb_sphere * sizeof(float4));
  data->sphere[0].x = 0;
  data->sphere[0].y = 0;
  data->sphere[0].z = 0;
  data->sphere[0].w = 1;
  data->meta.cam_pos.x = 0;
  data->meta.cam_pos.y = 5;
  data->meta.cam_pos.z = 0;
  data->meta.cam_rot.x = M_PI/2;
  data->meta.cam_rot.y = 0;
  data->meta.cam_rot.z = 0;
  data->meta.light.x = 5;
  data->meta.light.y = 5;
  data->meta.light.z = 5;
}

#include "hip/hip_runtime.h"
#include "core.h"

void init(s_data *data, SDL_Surface *screen)
{
  data->pixels = (unsigned int*)screen->pixels;
  data->nb_sphere = 1;
  data->sphere = (float4*)malloc(data->nb_sphere * sizeof(float4));
  data->sphere[0].x = 0;
  data->sphere[0].y = 0;
  data->sphere[0].z = 10;
  data->sphere[0].w = 1;
  data->cam_pos.x = 0;
  data->cam_pos.y = 0;
  data->cam_pos.z = 0;
  data->cam_rot.x = 0;
  data->cam_rot.y = 0;
  data->cam_rot.z = 0;
}

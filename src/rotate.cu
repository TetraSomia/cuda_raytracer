
#include <hip/hip_runtime.h>
__device__ void	rot_x(float3 *vec, float angle)
{
  float		tmp;

  tmp = vec->y;
  vec->y = tmp * cosf(angle) + vec->z * -sinf(angle);
  vec->z = tmp * sinf(angle) + vec->z * cosf(angle);
}

__device__ void    rot_y(float3 *vec, float angle)
{
  float		tmp;

  tmp = vec->x;
  vec->x = tmp * cosf(angle) + vec->z * sinf(angle);
  vec->z = tmp * -sinf(angle) + vec->z * cosf(angle);
}

__device__ void    rot_z(float3 *vec, float angle)
{
  float		tmp;

  tmp = vec->x;
  vec->x = tmp * cosf(angle) + vec->y * -sinf(angle);
  vec->y = tmp * sinf(angle) + vec->y * cosf(angle);
}

__device__ void	rot_vec(float3 *vec, float3 angle)
{
  rot_x(vec, angle.x);
  rot_y(vec, angle.y);
  rot_z(vec, angle.z);
}

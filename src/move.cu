#include "hip/hip_runtime.h"
#include <math.h>
#include "core.h"


int     key_forward(s_data *data)
{
  data->meta.cam_pos.x += MOVE * cos(data->meta.cam_rot.z);
  data->meta.cam_pos.y += MOVE * sin(data->meta.cam_rot.z);
  return (0);
}

int     key_backward(s_data *data)
{
  data->meta.cam_pos.x -= MOVE * cos(data->meta.cam_rot.z);
  data->meta.cam_pos.y -= MOVE * sin(data->meta.cam_rot.z);
  return (0);
}

int     key_left(s_data *data)
{
  data->meta.cam_pos.x += MOVE * cos(data->meta.cam_rot.z + M_PI / 2);
  data->meta.cam_pos.y += MOVE * sin(data->meta.cam_rot.z + M_PI / 2);
  return (0);
}

int     key_right(s_data *data)
{
  data->meta.cam_pos.x -= MOVE * cos(data->meta.cam_rot.z + M_PI / 2);
  data->meta.cam_pos.y -= MOVE * sin(data->meta.cam_rot.z + M_PI / 2);
  return (0);
}

int     key_up(s_data *data)
{
  data->meta.cam_pos.z += MOVE;
  return (0);
}

int     key_down(s_data *data)
{
  data->meta.cam_pos.z -= MOVE;
  return (0);
}

int	key_left_rotate(s_data *data)
{
  data->meta.cam_rot.z += ROT;
  return (0);
}

int	key_right_rotate(s_data *data)
{
  data->meta.cam_rot.z -= ROT;
  return (0);
}

int	key_up_rotate(s_data *data)
{
  data->meta.cam_rot.x -= ROT;
  return (0);
}

int	key_down_rotate(s_data *data)
{
  data->meta.cam_rot.x += ROT;
  return (0);
}


int key_listener(s_data *data)
{
  SDL_Event event;
  Uint8 *keystate = SDL_GetKeyState(NULL);

  if (keystate[SDLK_q])
    key_forward(data);
  if (keystate[SDLK_d])
    key_backward(data);
  if (keystate[SDLK_s])
    key_left(data);
  if (keystate[SDLK_z])
    key_right(data);
  if (keystate[SDLK_r])
    key_up(data);
  if (keystate[SDLK_f])
    key_down(data);
  if (keystate[SDLK_LEFT])
    key_left_rotate(data);
  if (keystate[SDLK_RIGHT])
    key_right_rotate(data);
  if (keystate[SDLK_UP])
    key_up_rotate(data);
  if (keystate[SDLK_DOWN])
    key_down_rotate(data);

  while (SDL_PollEvent(&event))
  {
    switch (event.type)
    {
    case SDL_KEYDOWN:
      break;
    case SDL_KEYUP:
      if (event.key.keysym.sym == SDLK_ESCAPE)
        return (1);
/*
      if (event.key.keysym.sym == SDLK_q)
        key_forward(data);
      if (event.key.keysym.sym == SDLK_d)
        key_backward(data);
      if (event.key.keysym.sym == SDLK_s)
        key_left(data);
      if (event.key.keysym.sym == SDLK_z)
        key_right(data);
*/
      break;
    case SDL_QUIT:
      return (1);
    }
  }
  return (0);
}

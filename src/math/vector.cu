#include "hip/hip_runtime.h"
#include <math.h>
#include "core.h"

__device__ float vec_dot(float3 v1, float3 v2)
{
  return (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z);
}

__device__ float3 vec_reflec(float3 n, float3 l)
{
  float	fac = 2.0f * vec_dot(n, l);

  return ((float3){fac * n.x - l.x, fac * n.y - l.y, fac * n.z - l.z});
}

__device__ float3 vec_new(float3 p1, float3 p2)
{
  return ((float3){p2.x - p1.x, p2.y - p1.y, p2.z - p1.z});
}

__device__ float3 vec_new_uni(float3 p1, float3 p2)
{
  float3 v;

  v.x = p2.x - p1.x;
  v.y = p2.y - p1.y;
  v.z = p2.z - p1.z;
  float mult = 1.0f / sqrtf(SQ(v.x) + SQ(v.y) + SQ(v.z));
  v.x *= mult;
  v.y *= mult;
  v.z *= mult;
  return (v);
}

__device__ float3 vec_inv(float3 v)
{
  return((float3){-v.x, -v.y, -v.z});
}

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "core.h"

SDL_Surface *screen;

void putpixel(int x, int y, int color)
{
  unsigned int *ptr = (unsigned int*)screen->pixels;
  int lineoffset = y * (screen->pitch / 4);
  ptr[lineoffset + x] = color;
}

void render(s_data *data)
{
  if (SDL_MUSTLOCK(screen))
    if (SDL_LockSurface(screen) < 0)
      return;
/*
  data->meta.cam_rot.x = data->rotation + M_PI/2;
  data->meta.cam_pos.y = 5*cos(data->rotation);
  data->meta.cam_pos.z = 5*sin(data->rotation);
  data->rotation += M_PI / 1024;
*/
  launch_kernel(data);

  if (SDL_MUSTLOCK(screen))
    SDL_UnlockSurface(screen);
  SDL_UpdateRect(screen, 0, 0, W_X, W_Y);
}

int main(int argc, char *argv[])
{
  s_data data;

  if (SDL_Init(SDL_INIT_VIDEO) < 0)
    exit(1);
  atexit(SDL_Quit);
  screen = SDL_SetVideoMode(W_X, W_Y, 32, SDL_SWSURFACE);// | SDL_FULLSCREEN);
  if (screen == NULL)
    exit(1);
  init(&data, screen);
  while (1)
  {
    render(&data);
    if (key_listener(&data) == 1)
      return (0);
  }
}

#include <stdlib.h>
#include "core.h"
#include "keys.h"

void render(s_data *data)
{
  if (SDL_MUSTLOCK(data->screen))
    if (SDL_LockSurface(data->screen) < 0)
      return;

  launch_kernel(data);

  if (SDL_MUSTLOCK(data->screen))
    SDL_UnlockSurface(data->screen);
  SDL_UpdateRect(data->screen, 0, 0, W_X, W_Y);
}

int main(int argc, char *argv[])
{
  s_data data;

  if (SDL_Init(SDL_INIT_VIDEO) < 0)
    exit(1);
  atexit(SDL_Quit);
  data.screen = SDL_SetVideoMode(W_X, W_Y, 32, SDL_SWSURFACE);// | SDL_FULLSCREEN);
  if (data.screen == NULL)
    exit(1);
  init(&data);
  while (1)
  {
    render(&data);
    if (key_listener(&data) == 1)
    {
      hipFree(data.g_sphere);
      hipFree(data.g_pixels);
      hipFree(data.g_meta);
      exit(0);
    }
  }
}

#include "hip/hip_runtime.h"
#include "core.h"

__device__ float solve_sphere_line_intersect(float3 p, float3 v, s_sphere sphere)
{
  float3 cst;
  float delta;
  float2 sol;

  cst.x = SQ(v.x) + SQ(v.y) + SQ(v.z);
  cst.y = 2.0f * (v.x * (p.x - sphere.pos.x)
	       + v.y * (p.y - sphere.pos.y)
	       + v.z * (p.z - sphere.pos.z));
  cst.z = -2.0f * (sphere.pos.x * p.x + sphere.pos.y * p.y
    + sphere.pos.z * p.z)
    + SQ(sphere.pos.x) + SQ(sphere.pos.y) + SQ(sphere.pos.z) - SQ(sphere.r)
    + SQ(p.x) + SQ(p.y) + SQ(p.z);
  delta = SQ(cst.y) - (4.0f * cst.x * cst.z);
  if (delta < 0.0f || cst.x == 0.0f)
    return (-1.0f);
  sol.x = (-(cst.y) + sqrt(delta)) / (2.0f * cst.x);
  sol.y = (-(cst.y) - sqrt(delta)) / (2.0f * cst.x);
  if (sol.x >= 0.0f && sol.y >= 0.0f)
    return (MIN(sol.x, sol.y));
  return (MAX(sol.x, sol.y));
}

__device__ void intersect(s_var *var, int nb_sphere, const s_sphere *sphere)
{
  float k;

  var->oc_d = INFINITE;
  for (int i = 0; i < nb_sphere; i++)
  {
    k = solve_sphere_line_intersect(var->c_p, var->c_v, sphere[i]);
    if (k >= 0.0f && k < var->oc_d)
    {
      var->oc_d = k;
      var->h_i = i;
    }
  }
  k = -var->c_p.z/var->c_v.z;
  if (k >= 0.0f && k < var->oc_d)
  {
    var->oc_d = k;
    var->h_i = -1;
  }
  if (var->oc_d == INFINITE)
    var->oc_d = -1.0f;
}

__device__ bool is_shadow(s_var *var, int nb_sphere, const s_sphere *sphere)
{
  float k;
  float k_min = INFINITE;
  int i_min = -1;
  float3 lo_v = vec_inv(var->ol_v);

  for (int i = 0; i < nb_sphere; i++)
  {
    k = solve_sphere_line_intersect(var->l_p, lo_v, sphere[i]);
    if (k >= 0.0f && k < k_min)
    {
      k_min = k;
      i_min = i;
    }
  }
  if (i_min < 0 && var->h_i >= 0)
    return (false);
  return (i_min ^ var->h_i);
}

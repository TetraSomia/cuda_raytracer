#include "hip/hip_runtime.h"
#include "core.h"

__device__ void init_ray(s_var *var, float3 cam_pos, float3 cam_rot, int i)
{
  var->c_p = cam_pos;
  var->c_v.x = (W_X / 2.0f) - (i % W_X);
  var->c_v.y = (W_Y / 2.0f) - (i / W_Y);
  var->c_v.z = sqrtf(SQ(W_X) + SQ(W_Y));
  rot_vec(&var->c_v, cam_rot);
}

__global__ void raytrace(float3 cam_pos, float3 cam_rot, uint nb_sphere, const float4 *sphere, uint *pixels)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  s_var var;

  init_ray(&var, cam_pos, cam_rot, i);
  intersect_spheres(&var, nb_sphere, sphere);
  pixels[i] = SET_REL_COLOR(var.oc_d, 50, 0);
}

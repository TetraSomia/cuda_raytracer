#include "hip/hip_runtime.h"
#include "core.h"

__device__ void init_ray(s_var *var, s_meta* meta, int i)
{
  var->l_p = meta->light;
  var->c_p = meta->cam_pos;
  var->c_v.x = (W_X / 2) - (i % W_X);
  var->c_v.y = (W_Y / 2) - (i / W_Y);
  var->c_v.z = sqrtf(SQ(W_X) + SQ(W_Y));
  rot_vec(&var->c_v, meta->cam_rot);
}

__global__ void raytrace(s_meta* meta, const s_sphere *sphere, uint *pixels)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  s_var var;

  // INTERSECT
  init_ray(&var, meta, i);
  intersect(&var, meta->nb_sphere, sphere);
  if (var.oc_d < 0.0f)
  {
    pixels[i] = 0;
    return;
  }

  // SHADERS
  light(&var, meta, sphere, &(pixels[i]));
  if (var.h_i >= 0 && !var.shadow)
    phong(&var);
}

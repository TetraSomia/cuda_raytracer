#include "hip/hip_runtime.h"
#include "core.h"

__device__ void get_hit_pos(s_var *var)
{
  var->h_p.x = var->c_p.x + var->oc_d * var->c_v.x;
  var->h_p.y = var->c_p.y + var->oc_d * var->c_v.y;
  var->h_p.z = var->c_p.z + var->oc_d * var->c_v.z;
}

__device__ void init_ray(s_var *var, s_meta* meta, int i)
{
  var->l_p = meta->light;
  var->c_p = meta->cam_pos;
  var->c_v.x = (W_X / 2.0f) - (i % W_X);
  var->c_v.y = (W_Y / 2.0f) - (i / W_Y);
  var->c_v.z = sqrtf(SQ(W_X) + SQ(W_Y));
  rot_vec(&var->c_v, meta->cam_rot);
}

__device__ inline float3 get_sphere_pos(float4 sphere)
{
  return ((float3){sphere.x, sphere.x, sphere.z});
}

__global__ void raytrace(s_meta* meta, const float4 *sphere, uint *pixels)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  s_var var;

  init_ray(&var, meta, i);
  intersect(&var, meta->nb_sphere, sphere);
  if (var.oc_d < 0)
  {
    pixels[i] = 0;
    return;
  }
  get_hit_pos(&var);
  if (var.h_i == -1)
  {
    var.h_n.x = 0;
    var.h_n.y = 0;
    var.h_n.z = 1;
  }
  else
  {
    var.o_p = get_sphere_pos(sphere[var.h_i]);
    var.h_n = vec_new_uni(var.o_p, var.h_p);
  }

  var.ol_v = vec_new_uni(var.h_p, var.l_p);
  var.coef = vec_dot(var.ol_v, var.h_n);

  if (var.coef < 0.1 || is_shadow(&var, meta->nb_sphere, sphere))
    var.coef = 0.1;

  var.coef *= 255.0f;
  pixels[i] = RGB((int)var.coef, (int)var.coef, (int)var.coef);
}

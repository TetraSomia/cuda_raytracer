#include "hip/hip_runtime.h"
#include "core.h"

__device__ void get_hit_pos(s_var *var)
{
  var->h_p.x = var->c_p.x + var->oc_d * var->c_v.x;
  var->h_p.y = var->c_p.y + var->oc_d * var->c_v.y;
  var->h_p.z = var->c_p.z + var->oc_d * var->c_v.z;
}

__device__ void init_ray(s_var *var, float3 cam_pos, float3 cam_rot, int i)
{
  var->c_p = cam_pos;
  var->c_v.x = (W_X / 2.0f) - (i % W_X);
  var->c_v.y = (W_Y / 2.0f) - (i / W_Y);
  var->c_v.z = sqrtf(SQ(W_X) + SQ(W_Y));
  rot_vec(&var->c_v, cam_rot);
}

__device__ inline float3 get_sphere_pos(float4 sphere)
{
  return ((float3){sphere.x, sphere.x, sphere.z});
}

__global__ void raytrace(s_meta* meta, const float4 *sphere, uint *pixels)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  s_var var;

  init_ray(&var, meta->cam_pos, meta->cam_rot, i);
  intersect_spheres(&var, meta->nb_sphere, sphere);
  if (var.oc_d < 0)
  {
    pixels[i] = 0;
    return;
  }
  get_hit_pos(&var);
  var.o_p = get_sphere_pos(sphere[var.h_i]);
  var.ol_v = vec_new_uni(var.h_p, meta->light);
  var.h_n = vec_new_uni(var.o_p, var.h_p);
  var.coef = vec_dot(var.ol_v, var.h_n);

  if (var.coef < 0)
    var.coef = 0;
  if (var.coef > 1)
    var.coef = 1;

  var.coef *= 255.0f;
  pixels[i] = RGB((int)var.coef, (int)var.coef, (int)var.coef);
}

#include "hip/hip_runtime.h"
#include <math.h>
#include "core.h"
#include "keys.h"

s_key_callback g_key_callback_tab[] =
{
  {SDLK_q, &key_forward},
  {SDLK_d, &key_backward},
  {SDLK_s, &key_left},
  {SDLK_z, &key_right},
  {SDLK_r, &key_up},
  {SDLK_f, &key_down},
  {SDLK_LEFT, &key_left_rotate},
  {SDLK_RIGHT, &key_right_rotate},
  {SDLK_UP, &key_up_rotate},
  {SDLK_DOWN, &key_down_rotate},
  {SDLK_UNKNOWN, NULL}
};

void key_forward(s_data *data)
{
  if (data->moved_object == CAMERA)
  {
    data->meta.cam_pos.x += MOVE * cos(data->meta.cam_rot.z);
    data->meta.cam_pos.y += MOVE * sin(data->meta.cam_rot.z);
  }
  else
  {
    data->sphere[data->moved_object].pos.x += MOVE * cos(data->meta.cam_rot.z);
    data->sphere[data->moved_object].pos.y += MOVE * sin(data->meta.cam_rot.z);
  }
}

void key_backward(s_data *data)
{
  if (data->moved_object == CAMERA)
  {
    data->meta.cam_pos.x -= MOVE * cos(data->meta.cam_rot.z);
    data->meta.cam_pos.y -= MOVE * sin(data->meta.cam_rot.z);
  }
  else
  {
    data->sphere[data->moved_object].pos.x -= MOVE * cos(data->meta.cam_rot.z);
    data->sphere[data->moved_object].pos.y -= MOVE * sin(data->meta.cam_rot.z);
  }
}

void key_left(s_data *data)
{
  if (data->moved_object == CAMERA)
  {
    data->meta.cam_pos.x += MOVE * cos(data->meta.cam_rot.z + M_PI / 2);
    data->meta.cam_pos.y += MOVE * sin(data->meta.cam_rot.z + M_PI / 2);
  }
  else
  {
    data->sphere[data->moved_object].pos.x += MOVE * cos(data->meta.cam_rot.z + M_PI / 2);
    data->sphere[data->moved_object].pos.y += MOVE * sin(data->meta.cam_rot.z + M_PI / 2);
  }
}

void key_right(s_data *data)
{
  if (data->moved_object == CAMERA)
  {
    data->meta.cam_pos.x -= MOVE * cos(data->meta.cam_rot.z + M_PI / 2);
    data->meta.cam_pos.y -= MOVE * sin(data->meta.cam_rot.z + M_PI / 2);
  }
  else
  {
    data->sphere[data->moved_object].pos.x -= MOVE * cos(data->meta.cam_rot.z + M_PI / 2);
    data->sphere[data->moved_object].pos.y -= MOVE * sin(data->meta.cam_rot.z + M_PI / 2);
  }
}

void key_up(s_data *data)
{
  if (data->moved_object == CAMERA)
    data->meta.cam_pos.z += MOVE;
  else
    data->sphere[data->moved_object].pos.z += MOVE;
}

void key_down(s_data *data)
{
  if (data->moved_object == CAMERA)
    data->meta.cam_pos.z -= MOVE;
  else
    data->sphere[data->moved_object].pos.z -= MOVE;
}

void key_left_rotate(s_data *data)
{
  data->meta.cam_rot.z += ROT;
}

void key_right_rotate(s_data *data)
{
  data->meta.cam_rot.z -= ROT;
}

void key_up_rotate(s_data *data)
{
  data->meta.cam_rot.x -= ROT;
}

void key_down_rotate(s_data *data)
{
  data->meta.cam_rot.x += ROT;
}
